#include "hip/hip_runtime.h"
#include "splatShader.h"
#include "config.h"
#include <hip/hip_cooperative_groups.h>
#ifndef GLM_FORCE_CUDA
#define GLM_FORCE_CUDA
#endif
#include <glm/glm.hpp>


namespace cg = cooperative_groups;

namespace SplatShader
{
    __device__ SplatShaderParams::SplatShaderParams(PackedSplatShaderParams p, int idx):
        W(p.W),
        H(p.H),
        time(p.time), dt(p.dt),
		position(p.positions[idx]),

        // world space position = 
        // view space up, right, forward,
        // world space up, right, forward,  		
		screen_position(p.screen_positions[idx]),
		viewmatrix(p.viewmatrix),
		viewmatrix_inv(p.viewmatrix_inv),
		projmatrix (p.projmatrix),
		projmatrix_inv (p.projmatrix_inv),
		focal_x (p.focal_x),
        focal_y (p.focal_y),
		tan_fovx (p.tan_fovx), 
        tan_fovy (p.tan_fovy),
        camera_position({p.viewmatrix_inv[12], p.viewmatrix_inv[13], p.viewmatrix_inv[14]}),

		// pr. frame texture information
		depth (p.depths[idx]),		
		conic_opacity (p.conic_opacity[idx]), // Todo: split opacity to own variable
        color_SH (p.colors_SH + idx),

		// Precomputed 'texture' information from the neilf pbr decomposition
		color_brdf ({p.features[idx * p.S + 0], p.features[idx * p.S + 1], p.features[idx * p.S + 2]}),
		normal ({p.features[idx * p.S + 3], p.features[idx * p.S + 4], p.features[idx * p.S + 5]}),
		color_base ({p.features[idx * p.S + 6], p.features[idx * p.S + 7], p.features[idx * p.S + 8]}),
		roughness (p.features[idx * p.S + 9]),
		metallic (p.features[idx * p.S + 10]),
		incident_light (p.features[idx * p.S + 11]),
		local_incident_light (p.features[idx * p.S + 12]),
		global_incident_light (p.features[idx * p.S + 13]),
		incident_visibility (p.features[idx * p.S + 14]),
        
        // Texture information
        d_textureManager(p.d_textureManager),

		// output
		// We use pointers to the output instead of return values to make it easy to extend during development.             
		out_color (p.out_colors + idx)
        {
		// for now we're not actually doing anyting in the constuctior other than initializing the constants.
    }

    __device__ static void DefaultSplatShaderCUDA(SplatShaderParams p)
    {
        // Set output color
        *p.out_color = (*p.color_SH);
    }

    __device__ static void OutlineShaderCUDA(SplatShaderParams p)
    {
        // Get angle between splat and camera:
        glm::vec3 directionToCamera = p.camera_position - p.position;
        float angle = 1 - glm::abs(glm::dot(glm::normalize(directionToCamera), glm::normalize(p.normal)));
        // easing from https://easings.net/#easeInOutQuint
        float opacity = angle < 0.5
            ? 1 - 16 * pow(angle, 5)
            : pow(-2 * angle + 2, 5) / 2;

        // Set output color
        *p.out_color = (*p.color_SH) * opacity;
    }

    __device__ static void WireframeShaderCUDA(SplatShaderParams p)
    {
        // Get angle between splat and camera:
        glm::vec3 directionToCamera = p.camera_position - p.position;
        float angle = 1 - glm::abs(glm::dot(glm::normalize(directionToCamera), glm::normalize(p.normal)));
        // easing from https://easings.net/#easeInOutQuint
        float opacity = angle < 0.5
            ? 1 - 16 * pow(angle, 5)
            : pow(-2 * angle + 2, 5) / 2;

        float rColor = fmodf(p.time / 5000, 1.0);
        // Set output color
        *p.out_color = glm::vec3(rColor, 1 - opacity,  1 - opacity);
    }

    __device__ static void TextureTestShaderCUDA(SplatShaderParams p)
    {

        char* texName = "Cracks";
        hipTextureObject_t crackTex = p.d_textureManager->GetTexture(texName);

        float4 sampleColor = tex2D<float4>(crackTex, p.position.x, p.position.y);
        
        *p.out_color = glm::vec3(sampleColor.x, sampleColor.y, sampleColor.z);
        
        //TODO: Make opacity something that can be modified in the shader.
        //p.conic_opacity.w = 1;
    }

    ///// Assign all the shaders to their short handles.
    // we need to keep them in constant device memory for them to stay valid when passed to host.
    __device__ const SplatShader defaultShader = &DefaultSplatShaderCUDA;
    __device__ const SplatShader outlineShader = &OutlineShaderCUDA;
    __device__ const SplatShader wireframeShader = &WireframeShaderCUDA;
    __device__ const SplatShader textureTestShader = &TextureTestShaderCUDA;


    std::map<std::string, int64_t> GetSplatShaderAddressMap(){
        // we cast pointers to numbers since most pointers aren't supported by pybind
        // Device function pointers seem to be 8 bytes long (at least on the devlopment machine with a GTX 2080 and when compiling to 64bit mode)
        // there doesn't seem to be a problem casting them to int64's though.

        std::map<std::string, int64_t> shaderMap;
        size_t shaderMemorySize = sizeof(SplatShader);
        
        // Copy device shader pointers to host map
        SplatShader::SplatShader h_defaultShader;
        hipMemcpyFromSymbol(&h_defaultShader, HIP_SYMBOL(defaultShader), shaderMemorySize);
        shaderMap["SplatDefault"] = (int64_t)h_defaultShader;

        SplatShader::SplatShader h_outlineShader;
        hipMemcpyFromSymbol(&h_outlineShader, HIP_SYMBOL(outlineShader), shaderMemorySize);
        shaderMap["OutlineShader"] = (int64_t)h_outlineShader;

        SplatShader::SplatShader h_wireframeShader;
        hipMemcpyFromSymbol(&h_wireframeShader, HIP_SYMBOL(wireframeShader), shaderMemorySize);
        shaderMap["WireframeShader"] = (int64_t)h_wireframeShader;

        SplatShader::SplatShader h_textureTestShader;
        hipMemcpyFromSymbol(&h_textureTestShader, HIP_SYMBOL(textureTestShader), shaderMemorySize);
        shaderMap["TextureTestShader"] = (int64_t)h_textureTestShader;

        return shaderMap;
    }

    // ALLOCATES THE RETURN ARRAY. REMEMBER TO FREE AFTER USE.
    // Returns an array in device memory containing addresses to device shader functions.
    int64_t* GetSplatShaderAddressArray(){
        // Array is assembled on CPU before being sent to device. Addresses themselves are in device space.
        int shaderCount = 4;
        int64_t* h_shaderArray = new int64_t[shaderCount];
        size_t shaderMemorySize = sizeof(SplatShader);
 
        SplatShader::SplatShader h_defaultShader;
        hipMemcpyFromSymbol(&h_defaultShader, HIP_SYMBOL(defaultShader), shaderMemorySize);
        h_shaderArray[0] = (int64_t)h_defaultShader;

        SplatShader::SplatShader h_outlineShader;
        hipMemcpyFromSymbol(&h_outlineShader, HIP_SYMBOL(outlineShader), shaderMemorySize);
        h_shaderArray[1] = (int64_t)h_outlineShader;

        SplatShader::SplatShader h_wireframeShader;
        hipMemcpyFromSymbol(&h_wireframeShader, HIP_SYMBOL(wireframeShader), shaderMemorySize);
        h_shaderArray[2] = (int64_t)h_wireframeShader;

        SplatShader::SplatShader h_textureTestShader;
        hipMemcpyFromSymbol(&h_textureTestShader, HIP_SYMBOL(textureTestShader), shaderMemorySize);
        h_shaderArray[3] = (int64_t)h_textureTestShader;

        // copy the host array to device
        int64_t* d_shaderArray;
        hipMalloc(&d_shaderArray, sizeof(int64_t)*shaderCount);
        hipMemcpy(d_shaderArray, h_shaderArray, shaderMemorySize * shaderCount, hipMemcpyDefault);

        // Delete temporary host array.
        delete[] h_shaderArray;
        return d_shaderArray;
    }


    __global__ void ExecuteShader(SplatShader* shaders, PackedSplatShaderParams packedParams){
        // calculate index for the spalt.
        auto idx = cg::this_grid().thread_rank();
        if (idx >= packedParams.P)
            return;

        // Unpack shader parameters into a format that is easier to work with. Increases memory footprint as tradeoff.
        // Could easily be optimized away by only indexing into the params inside the shader, but for now I'm prioritizing ease of use.
        SplatShaderParams params(packedParams, idx);

        // No need to dereference the shader function pointer.
        shaders[idx](params);
    }

}

