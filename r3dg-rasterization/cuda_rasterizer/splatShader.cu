#include "hip/hip_runtime.h"
#include "splatShader.h"
#include "config.h"
#include <hip/hip_cooperative_groups.h>


#ifndef GLM_FORCE_CUDA
#define GLM_FORCE_CUDA
#endif
#include <glm/glm.hpp>


namespace cg = cooperative_groups;

namespace SplatShader
{
    __device__ SplatShaderParams::SplatShaderParams(PackedSplatShaderParams p, int idx):
        W(p.W),
        H(p.H),
        time(p.time), dt(p.dt),
		position(p.positions[idx]),

        // world space position = 
        // view space up, right, forward,
        // world space up, right, forward,  		
		screen_position(p.screen_positions[idx]),
		viewmatrix(p.viewmatrix),
		viewmatrix_inv(p.viewmatrix_inv),
		projmatrix (p.projmatrix),
		projmatrix_inv (p.projmatrix_inv),
		focal_x (p.focal_x),
        focal_y (p.focal_y),
		tan_fovx (p.tan_fovx), 
        tan_fovy (p.tan_fovy),
        camera_position({p.viewmatrix_inv[12], p.viewmatrix_inv[13], p.viewmatrix_inv[14]}),

		// pr. frame texture information
		depth (p.depths[idx]),		
		conic_opacity (p.conic_opacity[idx]), // Todo: split opacity to own variable
        color_SH (p.colors_SH + idx),

		// Precomputed 'texture' information from the neilf pbr decomposition
		color_brdf ({p.features[idx * p.S + 0], p.features[idx * p.S + 1], p.features[idx * p.S + 2]}),
		normal ({p.features[idx * p.S + 3], p.features[idx * p.S + 4], p.features[idx * p.S + 5]}),
		color_base ({p.features[idx * p.S + 6], p.features[idx * p.S + 7], p.features[idx * p.S + 8]}),
		roughness (p.features[idx * p.S + 9]),
		metallic (p.features[idx * p.S + 10]),
		incident_light (p.features[idx * p.S + 11]),
		local_incident_light (p.features[idx * p.S + 12]),
		global_incident_light (p.features[idx * p.S + 13]),
		incident_visibility (p.features[idx * p.S + 14]),
        
        // Texture information
        d_textureManager(p.d_textureManager),

		// output
		// We use pointers to the output instead of return values to make it easy to extend during development.             
		out_color (p.out_colors + idx)
        {
		// for now we're not actually doing anyting in the constuctior other than initializing the constants.
    }

    __device__ static void DefaultSplatShaderCUDA(SplatShaderParams p)
    {
        // Set output color
        *p.out_color = (*p.color_SH);
    }

    __device__ static void OutlineShaderCUDA(SplatShaderParams p)
    {
        // Get angle between splat and camera:
        glm::vec3 directionToCamera = p.camera_position - p.position;
        float angle = 1 - glm::abs(glm::dot(glm::normalize(directionToCamera), glm::normalize(p.normal)));
        // easing from https://easings.net/#easeInOutQuint
        float opacity = angle < 0.5
            ? 1 - 16 * pow(angle, 5)
            : pow(-2 * angle + 2, 5) / 2;

        // Set output color
        *p.out_color = (*p.color_SH) * opacity;
    }

    __device__ static void WireframeShaderCUDA(SplatShaderParams p)
    {
        // Get angle between splat and camera:
        glm::vec3 directionToCamera = p.camera_position - p.position;
        float angle = 1 - glm::abs(glm::dot(glm::normalize(directionToCamera), glm::normalize(p.normal)));
        // easing from https://easings.net/#easeInOutQuint
        float opacity = angle < 0.5
            ? 1 - 16 * pow(angle, 5)
            : pow(-2 * angle + 2, 5) / 2;

        float rColor = fmodf(p.time / 5000, 1.0);
        // Set output color
        *p.out_color = glm::vec3(rColor, 1 - opacity,  1 - opacity);
    }

    __device__ static void TextureTestShaderCUDA(SplatShaderParams p)
    {

        char* texName = "Cracks";
        hipTextureObject_t crackTex = p.d_textureManager->GetTexture(texName);

        float4 sampleColor = tex2D<float4>(crackTex, p.position.x, p.position.y);
        
        *p.out_color = glm::vec3(sampleColor.x, sampleColor.y, sampleColor.z);

        //TODO: Make opacity something that can be modified in the shader.
        //p.conic_opacity.w = 1;
    }

    ///// Assign all the shaders to their short handles.
    // we need to keep them in constant device memory for them to stay valid when passed to host.
    __device__ const SplatShader defaultShader = &DefaultSplatShaderCUDA;
    __device__ const SplatShader outlineShader = &OutlineShaderCUDA;
    __device__ const SplatShader wireframeShader = &WireframeShaderCUDA;
    __device__ const SplatShader textureTestShader = &TextureTestShaderCUDA;


    IndirectMap<char*, SplatShader>* GetSplatShaderAddressMap(){
        std::vector<char*> shaderNames;
        std::vector<SplatShader> shaderFunctionPointers;
        size_t shaderMemorySize = sizeof(SplatShader);
        
        // Copy device shader pointers to host, so we can store them in the indirect map.
        SplatShader::SplatShader h_defaultShader;
        hipMemcpyFromSymbol(&h_defaultShader, HIP_SYMBOL(defaultShader), shaderMemorySize);
        shaderFunctionPointers.push_back(h_defaultShader);
        shaderNames.push_back("SplatDefault");

        SplatShader::SplatShader h_outlineShader;
        hipMemcpyFromSymbol(&h_outlineShader, HIP_SYMBOL(outlineShader), shaderMemorySize);
        shaderFunctionPointers.push_back(h_outlineShader);
        shaderNames.push_back("OutlineShader");

        SplatShader::SplatShader h_wireframeShader;
        hipMemcpyFromSymbol(&h_wireframeShader, HIP_SYMBOL(wireframeShader), shaderMemorySize);
        shaderFunctionPointers.push_back(h_wireframeShader);
        shaderNames.push_back("WireframeShader");

        SplatShader::SplatShader h_textureTestShader;
        hipMemcpyFromSymbol(&h_textureTestShader, HIP_SYMBOL(textureTestShader), shaderMemorySize);
        shaderFunctionPointers.push_back(h_textureTestShader);
        shaderNames.push_back("TextureTestShader");

        IndirectMap<char*, SplatShader>* splatShaderMap = new IndirectMap<char*, SplatShader>(shaderNames, shaderFunctionPointers);

        return splatShaderMap;
    }

    __global__ void ExecuteShader(SplatShader* shaders, PackedSplatShaderParams packedParams){
        // calculate index for the spalt.
        auto idx = cg::this_grid().thread_rank();
        if (idx >= packedParams.P)
            return;

        // Unpack shader parameters into a format that is easier to work with. Increases memory footprint as tradeoff.
        // Could easily be optimized away by only indexing into the params inside the shader, but for now I'm prioritizing ease of use.
        SplatShaderParams params(packedParams, idx);

        // No need to dereference the shader function pointer.
        shaders[idx](params);
    }

}

