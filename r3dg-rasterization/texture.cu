#include "hip/hip_runtime.h"
#include "texture.h"
#include "cuda_rasterizer/auxiliary.h"
#include <iostream>
#include <stdexcept>
#include <map>
#include <hip/hip_cooperative_groups.h>
#include "charOperations.cu"


namespace Texture
{
    // Allocates a new array from the input array where every 4th index is a padded value of 1. The input pointer is overwritten with the pointer to the new array.
    // The array is 4/3rds the length of the input array. Remember to delete the allocated array.
    __global__ void CreatPaddedArrayFromBase(float* src, float* dest, int paddedDataCount){
        auto padded_i = cooperative_groups::this_grid().thread_rank();
        if (padded_i >= paddedDataCount)
            return;

        bool padCurrentIndex = (padded_i + 1) % 4 == 0;
        if (padCurrentIndex)
        {
            dest[padded_i] = 1;
        }
        else{
            // paddedDataCount = height * width * 4
            // srcDataCount = height * width * 3
            int src_i = ceil(padded_i*0.75);
            dest[padded_i] = src[src_i];
        }
    }

    // Encodes a string representing a texture mode to a TextureMode enum
    int EncodeTextureMode(std::string mode){
        if(mode == "1")
            return TextureMode::One;
        else if ( mode == "L")
            return TextureMode::L;
        else if ( mode == "P")
            return TextureMode::P;
        else if ( mode == "RGB")
            return TextureMode::RGB;
        else if ( mode == "RGBA")
            return TextureMode::RGBA;
        else if ( mode == "CMYK")
            return TextureMode::CMYK;
        else if ( mode == "YCbCr")
            return TextureMode::YCbCr;
        else if ( mode == "LAB")
            return TextureMode::LAB;
        else if ( mode == "HSV")
            return TextureMode::HSV;
        else if ( mode == "I")
            return TextureMode::I;
        else if ( mode == "F")
            return TextureMode::F;

        return TextureMode::Unknown;
    }

    // Encodes string to cuda enum. Possible modes are:
    // - "Wrap": UVs outside the range wraps back around from the other side, repeating the texture.
    // - "Mirror": UBs outside the range wraps back from the same side, repeating the texture, but mirrored.
    // - "Clamp": UVs outside the range are clamped back into the range of the texture
    // - "Border": UVs outside the range return 0.
    int EncodeWrapMode(std::string mode){
        if(mode == "Border")
            return hipAddressModeBorder;
        if(mode == "Clamp")
            return hipAddressModeClamp;
        if(mode == "Mirror")
            return hipAddressModeMirror;
        if(mode == "Wrap")
            return hipAddressModeWrap;

        return -1;
    }

    // NOTICE: Returns a hipTextureObject_t* cast to a int64_t
    // Creates a textureObject wrapper around the provided texture data and allocates it in memory.
    // Used in order to intialize textures outside of render loop.
    int64_t AllocateTexture(std::map<std::string, torch::Tensor> textureData){
        hipTextureObject_t* texObj = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t));

        CreateTexture(texObj, textureData);

        printf("C++ pointer saved at %A\n", texObj);

        return (int64_t)texObj;
    }

    // Creates a textureObject wrapper around the provided texture data
    // Adapted from the cuda example at https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api
    void CreateTexture(hipTextureObject_t* texObjPtr, std::map<std::string, torch::Tensor> textureData){
        // extract all the texture data
        int height = textureData["height"].const_data_ptr<int>()[0];
        int width = textureData["width"].const_data_ptr<int>()[0];
        TextureMode mode = static_cast<TextureMode>(textureData["encoding_mode"].const_data_ptr<int>()[0]);
        float* pixelData = textureData["pixelData"].contiguous().cuda().mutable_data_ptr<float>();
        int pixelDataCount = textureData["pixelData"].numel();
        const hipTextureAddressMode* addressModes = (const hipTextureAddressMode*)textureData["wrap_modes"].const_data_ptr<int>();

        /*
        Create a channel descriptor appropriate to the image mode.
        Pillow supported channel modes:
        1 (1-bit pixels, black and white, stored with one pixel per byte)
        L (8-bit pixels, grayscale)
        P (8-bit pixels, mapped to any other mode using a color palette)
        RGB (3x8-bit pixels, true color)
        RGBA (4x8-bit pixels, true color with transparency mask)
        CMYK (4x8-bit pixels, color separation)
        YCbCr (3x8-bit pixels, color video format)
        Note that this refers to the JPEG, and not the ITU-R BT.2020, standard
        LAB (3x8-bit pixels, the L*a*b color space)
        HSV (3x8-bit pixels, Hue, Saturation, Value color space)
        Hue’s range of 0-255 is a scaled version of 0 degrees <= Hue < 360 degrees
        I (32-bit signed integer pixels)
        F (32-bit floating point pixels)
        https://pillow.readthedocs.io/en/stable/handbook/concepts.html
        */
        hipChannelFormatDesc channelDesc;
        int paddedDataSize;
        int channelByteWidth;
        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        if(mode == TextureMode::One || mode == TextureMode::L || mode == TextureMode::P || mode == TextureMode::I || mode == TextureMode::F){
            channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float1);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGBA || mode == TextureMode::CMYK)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3, so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        else if (mode == TextureMode::LAB|| mode == TextureMode::HSV)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModePoint;  // Linear filtering is only available for floats 
            texDesc.readMode = hipReadModeNormalizedFloat; //Notice: This might actually not work with 32 bit channels. Only 16 and 8 bit channels. But I haven't tested it.

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3m so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        // Specify remaining texture object parameters
        texDesc.addressMode[0] = addressModes[0];
        texDesc.addressMode[1] = addressModes[1];
        texDesc.normalizedCoords = 1;                           //TODO: allow the coordinate mode ot be set based on a texture import setting

        // Allocate CUDA array in device memory
        hipArray_t cuArray;
        hipMallocArray(&cuArray, &channelDesc, width, height);

        // Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src, including padding). We dont have any padding, so it's just equal to the byte width.
        const size_t pitch = channelByteWidth;
        checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, pixelData, pitch, channelByteWidth, height, hipMemcpyDeviceToDevice));

        // Specify texture
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;
        
        // Create texture object, which is used as a wrapper to access the cuda Array with the actual image data.
        checkCudaErrors(hipCreateTextureObject(texObjPtr, &resDesc, &texDesc, NULL));

        if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr || mode == TextureMode::LAB || mode == TextureMode::HSV)
        {   
            // If we had to copy and pad the data of a 3-value format with a 4th value before the data was copied to a hipArray,
            // we have to free the memory used to create the temporary padded version. 
            hipFree(pixelData);
        }
    }

    // NOTICE: Returns a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* cast to an int64_t in order to get around the pybind pointer wierdness.
    // Takes the texture tensor Maps and uses it to create wrapper objects around the texture data, so it can be accessed efficiently in the shaders.
    // shaderTextureTensorMaps stores data in nested maps on the format: <ShaderName, <TextureName, <TexturePropertyName, TexturePropertyData*>>>
	// shaderTextureMaps stores data in nested maps on the format: <ShaderName, <TextureName, TextureObject*>>
    int64_t InitializeTextureMaps(
        const std::map<std::string, std::map<std::string, std::map<std::string, torch::Tensor>>>& shaderTextureTensorMaps)
    {
        auto shaderTextureMaps = new std::map<std::string, std::map<std::string, hipTextureObject_t*>>;

        for(auto shaderTextureTensorBundle : shaderTextureTensorMaps){
            std::string shaderName = shaderTextureTensorBundle.first;
            auto textureTensorBundle = shaderTextureTensorBundle.second;

            for(auto textureTensor : textureTensorBundle){
                std::string textureName = textureTensor.first;
                auto textureData = textureTensor.second;

                hipTextureObject_t* texObj = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t)); // TODO: Does this actually need to be malloced? 
                Texture::CreateTexture(texObj, textureData);
                (*shaderTextureMaps)[shaderName][textureName] = texObj;
            }
        }

        //std::cout << "ShaderTextureBundle poiter" << shaderTextureMaps << ". Casting to " << (int64_t)shaderTextureMaps << std::endl;
        return (int64_t)shaderTextureMaps;
    }

    // NOTICE: Only call if the pointer haven't been passed through python. See InitializeTextureWrappers() comment.
    // Frees the underlying hipArray that the textureObject is wrapped around, as well as the texture object pointer that contains it.
    void UnloadTexture(hipTextureObject_t* textureObject){
        hipResourceDesc resDesc;
        checkCudaErrors(hipGetTextureObjectResourceDesc(&resDesc, (*textureObject)));
        checkCudaErrors(hipFreeArray(resDesc.res.array.array));
        delete(textureObject);
    }

    // NOTICE: takes a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* that has been cast to an int64_t in order to get around the pybind pointer wierdness.
    // Unloads all the memory allocated for all the texture Maps, including the input pointer.
    void UnloadTextureMaps (int64_t shaderTextureMaps_mapPtr){
        auto shaderTextureMaps = (std::map<std::string, std::map<std::string, hipTextureObject_t*>>*)shaderTextureMaps_mapPtr;

        // For each shader, unload all textures from memory
        for(auto shaderTextureBundle : (*shaderTextureMaps)){
            auto textureBundle = shaderTextureBundle.second;
            
            for(auto texture : textureBundle){
                hipTextureObject_t* texObj = texture.second;
                Texture::UnloadTexture(texObj);
            }
        }

        delete(shaderTextureMaps);
    }


    // initialize device texture vector and device texture name vector (used for indirect addressing of textures)
    // NOTICE: actually returns a std::pair<char**,hipTextureObject_t*> cast to a pair of int64s.
    std::pair<int64_t, int64_t> LoadDeviceTextureLookupTable(std::vector<std::string> names, std::vector<int64_t> textureObjects){
    
        // First, move each element into a vector on host
        // We use vectors instead of arrays since we don't know the size at compile time, and we don't wanna allocate memory ourselves.
        std::vector<char*> h_names (names.size());
        std::vector<hipTextureObject_t*> h_texObjs(names.size());

        for (size_t i = 0; i < names.size(); i++)
        {
            std::string name = names[i];
            hipTextureObject_t* texObj = (hipTextureObject_t*)textureObjects[i];
            printf("%s C++ pointer saved at %A\n", name.c_str(), texObj);

            // convert each name to a char array located in device memory
            // Texture objects are already in device memory, so we don't need to do anything to them.
            int stringlength = name.length();
            char* d_charName;
            hipMalloc(&d_charName, (stringlength+1)*sizeof(char)); // add 1 to also include the termination character
            hipMemcpy(d_charName, name.c_str(), stringlength+1, hipMemcpyKind::hipMemcpyDefault);

            // Save the pointers to the tempoary host vectors.
            h_names[i] = d_charName;
            h_texObjs[i] = texObj;
        }

        // Then create a couple of device arrays and transfer all data to them.
        // We have to allocate new memory for the device vectors, so that they stay persitant over multiple render loops.
        char** d_names;
        hipMalloc(&d_names, names.size() * sizeof(char*));
        hipMemcpy(d_names, &(h_names[0]), h_names.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipTextureObject_t* d_texObjs;
        hipMalloc(&d_texObjs, names.size() * sizeof(hipTextureObject_t*));
        hipMemcpy(d_texObjs, &(h_texObjs[0]), h_texObjs.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        return std::pair((int64_t)d_names, (int64_t)d_texObjs);
    }


    // --------------- Debug methods ---------------

    void TestFunctionPointerMap(){
        int N = 100;
        thrust::device_vector<int> d_a(N);
    }

    // Test whether we can do the texture initialization before the call.
    // Returns an int* cast to an int64_t in order to get around pybind wierdness.
    int64_t AllocateVariable(){
        int* allocedPointer = (int*) malloc(sizeof(int)); 
        (*allocedPointer) = 10;
        std::cout << "C++ pointer saved at " << allocedPointer << std::endl;
        return (int64_t)allocedPointer;
    }

    void PrintVariable (int64_t allocedPointer_intPtr){
        std::cout << "Reading following value from alloced pointer: " << (*((int*)allocedPointer_intPtr)) << std::endl;
    }

    void DeleteVariable(int64_t allocatedPointer_intPtr){
        std::cout << "Deleting allocated pointer" << std::endl;
        delete (int*)allocatedPointer_intPtr;
        std::cout << "Deleting done" << std::endl;
    }

    // NOTICE: takes a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* cast to an int64_t in order to get around the pybind pointer wierdness.
    void PrintFromFirstTexture (int64_t shaderTextureMaps_Ptr){
        auto shaderTextureMaps = (std::map<std::string, std::map<std::string, hipTextureObject_t*>>*)shaderTextureMaps_Ptr;
        //std::cout << "Cast shaderTextureBundle map pointer from" << shaderTextureMaps_Ptr << " back to " << shaderTextureMaps << std::endl;

        // For each shader, unload all textures from memory
        for(auto shaderTextureBundle : (*shaderTextureMaps)){
            auto textureBundle = shaderTextureBundle.second;

            for(auto texture : textureBundle){
                hipTextureObject_t* texObj = texture.second;
                PrintFirstPixel<<<1,1>>>((*texObj));
                hipDeviceSynchronize();
            }
        }
    }


    void PrintFromWrappedTexture(int64_t texObj_int64_t_ptr){
        hipTextureObject_t* texObj = (hipTextureObject_t*)texObj_int64_t_ptr;
        PrintFirstPixel<<<1,1>>>((*texObj));
        hipDeviceSynchronize();
    }


    __device__ void PrintFromTextureLookuptableCUDA(char** texNames, hipTextureObject_t* texObjs, int texCount, const char* targetName){
        // Find the target texture name index:
        for (size_t i = 0; i < texCount; i++)
        {
            char* name = texNames[i];

            // Check if the name in the lookup table is the same as the target name
            bool textureHasBeenFound = charsAreEqual(name, targetName);

            // if so, print the pixel value and return
            if(textureHasBeenFound){
                hipTextureObject_t texObj = texObjs[i];
                printf("C++ pointer saved at %A\n", &texObjs[i]);
                PrintFirstPixel<<<1,1>>>(texObj);
                hipDeviceSynchronize();
                return;
            }
        }
        
        // If none of the textures had the correct name, print a warning:
        printf("Warning: Texture '%s' not found\n", targetName);
        hipDeviceSynchronize();
    }

    
    void PrintFromTextureLookuptable(std::pair<int64_t, int64_t> texLookupTable, int texCount, std::string targetName){
        auto texNames = (char**)texLookupTable.first;
        auto texObjs = (hipTextureObject_t*)texLookupTable.second;
        PrintFromTextureLookuptableCUDA<<<1,1>>>(texNames, texObjs, texCount, targetName.c_str());
    }


    // Debug Method used for quickly testing whether 
    __global__ void PrintFirstPixel(hipTextureObject_t texObj){
        float4 cudaTexel = tex2D<float4>(texObj, 0, 0);
        printf("Cuda reading RGBA value of first texel: %f,%f,%f,%f\n", cudaTexel.x, cudaTexel.y, cudaTexel.z, cudaTexel.w);
    }
}