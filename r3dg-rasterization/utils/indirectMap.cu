#include "indirectMap.h"
#include "charOperations.h"

template <typename KeyType, typename ValueType>
__host__ IndirectMap<KeyType, ValueType>::IndirectMap(){
    // Pass for now
};

template <typename ValueType>
__host__ IndirectMap<char*, ValueType>::IndirectMap(){
    // Pass for now
};

template <typename KeyType, typename ValueType>
__host__ IndirectMap<KeyType, ValueType>::IndirectMap(std::vector<KeyType> keys, std::vector<ValueType> values){
    SetAll(keys, values);
};

template <typename ValueType>
__host__ IndirectMap<char*, ValueType>::IndirectMap(std::vector<char*> keys, std::vector<ValueType> values){
    SetAll(keys, values);
};

// Allocates and Uploads keys and values to internal device arrays.
template <typename KeyType, typename ValueType>
__host__ void IndirectMap<KeyType, ValueType>::SetAll(std::vector<KeyType> keys, std::vector<ValueType> values){
    int h_keyCount = keys.size();
    hipMalloc(&d_itemCount, sizeof(int));
    hipMemcpy(d_itemCount, &h_keyCount, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_keys, h_keyCount * sizeof(KeyType));
    hipMemcpy(d_keys, &keys[0], h_keyCount *sizeof(KeyType), hipMemcpyHostToDevice);

    hipMalloc(&d_values, h_keyCount * sizeof(ValueType));
    hipMemcpy(d_values, &values[0], h_keyCount * sizeof(ValueType), hipMemcpyKind::hipMemcpyDefault);
}

// Allocates and Uploads keys and values to internal device arrays.
template <typename ValueType>
__host__ void IndirectMap<char*, ValueType>::SetAll(std::vector<char*> keys, std::vector<ValueType> values){
        int h_keyCount = keys.size();
        std::vector<char*> h_keys (h_keyCount);

        // First we have to transfer each char array individually to the GPU.
        for (size_t i = 0; i < h_keyCount; i++)
        {
            std::string name = names[i];

            int stringlength = name.length();
            char* d_charName;
            hipMalloc(&d_charName, (stringlength+1)*sizeof(char)); // add 1 to also include the termination character
            hipMemcpy(d_charName, name.c_str(), stringlength+1, hipMemcpyKind::hipMemcpyDefault);

            // Save the pointers to the tempoary host vectors.
            h_keys[i] = d_charName;
        }

        // Then allocate all the arrays on the device and transfer the data stored on host to them (and to the texCount variable):
        hipMalloc(&this->d_itemCount, sizeof(int));
        hipMemcpy(this->d_itemCount, &h_keyCount, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&this->d_keys, h_keyCount * sizeof(char*));
        hipMemcpy(this->d_keys, &h_keys[0], h_keyCount *sizeof(char*), hipMemcpyHostToDevice);

        hipMalloc(&this->d_values, h_keyCount * sizeof(ValueType));
        hipMemcpy(this->d_values, &values[0], h_keyCount * sizeof(ValueType), hipMemcpyKind::hipMemcpyDefault);
}

//Read from map on device
template <typename KeyType, typename ValueType>
__device__ ValueType IndirectMap<KeyType, ValueType>::operator[](KeyType key){
    for (size_t i = 0; i < *this->d_itemCount; i++)
    {
        // Check if the name in the lookup table is the same as the target name
        KeyType currentKey = this->d_keys[i];
        bool keyHasBeenFound = key == currentKey;
        
        if(keyHasBeenFound){
            ValueType value = this->d_values[i];
            return value;
        }
    }

    // If the value can't be found, we would usually throw an error, but we can't throw errors in cuda.
    // So instead we assert a false statement to make the program halt.
    assert(false);
}

//Read from map on device
template <typename KeyType, typename ValueType>
__device__ ValueType IndirectMap<KeyType, ValueType>::Get(KeyType key){
    return this[key];
}

//Read from map on device. As a special case, if the key type is a char*, we will try to compare the contents of the char array
//instead of the actual pointers.
template <typename ValueType>
__device__ ValueType IndirectMap<char*, ValueType>::operator[](char* string){
    for (size_t i = 0; i < *this->d_itemCount; i++)
    {
        // Check if the name in the lookup table is the same as the target name
        char* currentKey = this->d_keys[i];
        bool keyHasBeenFound = charsAreEqual(key, currentKey);
        
        if(textureHasBeenFound){
            ValueType value = this->d_values[i];
            return value;
        }
    }

    // If the value can't be found, we would usually throw an error, but we can't throw errors in cuda.
    // So instead we assert a false statement to make the program halt.
    assert(false);
}

template <typename ValueType>
__device__ ValueType IndirectMap<char*, ValueType>::Get(char* string){
    return this[string];
}

template <typename KeyType, typename ValueType>
__device__ IndirectMap<KeyType, ValueType>::~IndirectMap(){
    hipFree(this->d_itemCount);
    hipFree(this->d_keys);
    hipFree(this->d_values);
}

template <typename ValueType>
__device__ IndirectMap<char*, ValueType>::~IndirectMap(){
    hipFree(this->d_itemCount);
    hipFree(this->d_keys);
    hipFree(this->d_values);
}