#include "hip/hip_runtime.h"
#include "texture.h"
#include "cuda_rasterizer/auxiliary.h"
#include <iostream>
#include <stdexcept>
#include <map>
#include <hip/hip_cooperative_groups.h>

namespace Texture
{
    // Allocates a new array from the input array where every 4th index is a padded value of 1. The input pointer is overwritten with the pointer to the new array.
    // The array is 4/3rds the length of the input array. Remember to delete the allocated array.
    __global__ void CreatPaddedArrayFromBase(float* src, float* dest, int paddedDataCount){
        auto padded_i = cooperative_groups::this_grid().thread_rank();
        if (padded_i >= paddedDataCount)
            return;

        bool padCurrentIndex = (padded_i + 1) % 4 == 0;
        if (padCurrentIndex)
        {
            dest[padded_i] = 1;
        }
        else{
            // paddedDataCount = height * width * 4
            // srcDataCount = height * width * 3
            int src_i = ceil(padded_i*0.75);
            dest[padded_i] = src[src_i];
        }
    }

    // Encodes a string representing a texture mode to a TextureMode enum
    int EncodeTextureMode(std::string mode){
        if(mode == "1")
            return TextureMode::One;
        else if ( mode == "L")
            return TextureMode::L;
        else if ( mode == "P")
            return TextureMode::P;
        else if ( mode == "RGB")
            return TextureMode::RGB;
        else if ( mode == "RGBA")
            return TextureMode::RGBA;
        else if ( mode == "CMYK")
            return TextureMode::CMYK;
        else if ( mode == "YCbCr")
            return TextureMode::YCbCr;
        else if ( mode == "LAB")
            return TextureMode::LAB;
        else if ( mode == "HSV")
            return TextureMode::HSV;
        else if ( mode == "I")
            return TextureMode::I;
        else if ( mode == "F")
            return TextureMode::F;

        return TextureMode::Unknown;
    }

    // Encodes string to cuda enum. Possible modes are:
    // - "Wrap": UVs outside the range wraps back around from the other side, repeating the texture.
    // - "Mirror": UBs outside the range wraps back from the same side, repeating the texture, but mirrored.
    // - "Clamp": UVs outside the range are clamped back into the range of the texture
    // - "Border": UVs outside the range return 0.
    int EncodeWrapMode(std::string mode){
        if(mode == "Border")
            return hipAddressModeBorder;
        if(mode == "Clamp")
            return hipAddressModeClamp;
        if(mode == "Mirror")
            return hipAddressModeMirror;
        if(mode == "Wrap")
            return hipAddressModeWrap;

        return -1;
    }

    // Creates a textureObject wrapper around the provided texture data
    // Adapted from the cuda example at https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api
    void CreateTexture(hipTextureObject_t* texObjPtr, std::map<std::string, torch::Tensor> textureData){
        // extract all the texture data
        int height = textureData["height"].const_data_ptr<int>()[0];
        int width = textureData["width"].const_data_ptr<int>()[0];
        TextureMode mode = static_cast<TextureMode>(textureData["encoding_mode"].const_data_ptr<int>()[0]);
        float* pixelData = textureData["pixelData"].contiguous().cuda().mutable_data_ptr<float>();
        int pixelDataCount = textureData["pixelData"].numel();
        const hipTextureAddressMode* addressModes = (const hipTextureAddressMode*)textureData["wrap_modes"].const_data_ptr<int>();

        /*
        Create a channel descriptor appropriate to the image mode.
        Pillow supported channel modes:
        1 (1-bit pixels, black and white, stored with one pixel per byte)
        L (8-bit pixels, grayscale)
        P (8-bit pixels, mapped to any other mode using a color palette)
        RGB (3x8-bit pixels, true color)
        RGBA (4x8-bit pixels, true color with transparency mask)
        CMYK (4x8-bit pixels, color separation)
        YCbCr (3x8-bit pixels, color video format)
        Note that this refers to the JPEG, and not the ITU-R BT.2020, standard
        LAB (3x8-bit pixels, the L*a*b color space)
        HSV (3x8-bit pixels, Hue, Saturation, Value color space)
        Hue’s range of 0-255 is a scaled version of 0 degrees <= Hue < 360 degrees
        I (32-bit signed integer pixels)
        F (32-bit floating point pixels)
        https://pillow.readthedocs.io/en/stable/handbook/concepts.html
        */
        hipChannelFormatDesc channelDesc;
        int paddedDataSize;
        int channelByteWidth;
        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        if(mode == TextureMode::One || mode == TextureMode::L || mode == TextureMode::P || mode == TextureMode::I || mode == TextureMode::F){
            channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float1);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGBA || mode == TextureMode::CMYK)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3, so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        else if (mode == TextureMode::LAB|| mode == TextureMode::HSV)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModePoint;  // Linear filtering is only available for floats 
            texDesc.readMode = hipReadModeNormalizedFloat; //Notice: This might actually not work with 32 bit channels. Only 16 and 8 bit channels. But I haven't tested it.

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3m so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        // Specify remaining texture object parameters
        texDesc.addressMode[0] = addressModes[0];
        texDesc.addressMode[1] = addressModes[1];
        texDesc.normalizedCoords = 1;                           //TODO: allow the coordinate mode ot be set based on a texture import setting

        // Allocate CUDA array in device memory
        hipArray_t cuArray;
        hipMallocArray(&cuArray, &channelDesc, width, height);

        // Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src, including padding). We dont have any padding, so it's just equal to the byte width.
        const size_t pitch = channelByteWidth;
        checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, pixelData, pitch, channelByteWidth, height, hipMemcpyDeviceToDevice));

        // Specify texture
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;
        
        // Create texture object, which is used as a wrapper to access the cuda Array with the actual image data.
        checkCudaErrors(hipCreateTextureObject(texObjPtr, &resDesc, &texDesc, NULL));

        if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr || mode == TextureMode::LAB || mode == TextureMode::HSV)
        {   
            // If we had to copy and pad the data of a 3-value format with a 4th value before the data was copied to a hipArray,
            // we have to free the memory used to create the temporary padded version. 
            hipFree(pixelData);
        }
    }

    // NOTICE: Returns a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* cast to an int64_t in order to get around the pybind pointer wierdness.
    // Takes the texture tensor Maps and uses it to create wrapper objects around the texture data, so it can be accessed efficiently in the shaders.
    // shaderTextureTensorMaps stores data in nested maps on the format: <ShaderName, <TextureName, <TexturePropertyName, TexturePropertyData*>>>
	// shaderTextureMaps stores data in nested maps on the format: <ShaderName, <TextureName, TextureObject*>>
    int64_t InitializeTextureMaps(
        const std::map<std::string, std::map<std::string, std::map<std::string, torch::Tensor>>>& shaderTextureTensorMaps)
    {
        auto shaderTextureMaps = new std::map<std::string, std::map<std::string, hipTextureObject_t*>>;

        for(auto shaderTextureTensorBundle : shaderTextureTensorMaps){
            std::string shaderName = shaderTextureTensorBundle.first;
            auto textureTensorBundle = shaderTextureTensorBundle.second;

            for(auto textureTensor : textureTensorBundle){
                std::string textureName = textureTensor.first;
                auto textureData = textureTensor.second;

                hipTextureObject_t* texObj = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t)); // TODO: Does this actually need to be malloced? 
                Texture::CreateTexture(texObj, textureData);
                (*shaderTextureMaps)[shaderName][textureName] = texObj;
            }
        }

        //std::cout << "ShaderTextureBundle poiter" << shaderTextureMaps << ". Casting to " << (int64_t)shaderTextureMaps << std::endl;
        return (int64_t)shaderTextureMaps;
    }

    // NOTICE: Only call if the pointer haven't been passed through python. See InitializeTextureWrappers() comment.
    // Frees the underlying hipArray that the textureObject is wrapped around, as well as the texture object pointer that contains it.
    void UnloadTexture(hipTextureObject_t* textureObject){
        hipResourceDesc resDesc;
        checkCudaErrors(hipGetTextureObjectResourceDesc(&resDesc, (*textureObject)));
        checkCudaErrors(hipFreeArray(resDesc.res.array.array));
        delete(textureObject);
    }

    // NOTICE: takes a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* that has been cast to an int64_t in order to get around the pybind pointer wierdness.
    // Unloads all the memory allocated for all the texture Maps, including the input pointer.
    void UnloadTextureMaps (int64_t shaderTextureMaps_mapPtr){
        auto shaderTextureMaps = (std::map<std::string, std::map<std::string, hipTextureObject_t*>>*)shaderTextureMaps_mapPtr;

        // For each shader, unload all textures from memory
        for(auto shaderTextureBundle : (*shaderTextureMaps)){
            auto textureBundle = shaderTextureBundle.second;
            
            for(auto texture : textureBundle){
                hipTextureObject_t* texObj = texture.second;
                Texture::UnloadTexture(texObj);
            }
        }

        delete(shaderTextureMaps);
    }

    // --------------- Debug methods ---------------

    // Test whether we can do the texture initialization before the call.
    // Returns an int* cast to an int64_t in order to get around pybind wierdness.
    int64_t AllocateVariable(){
        int* allocedPointer = (int*) malloc(sizeof(int)); 
        (*allocedPointer) = 10;
        std::cout << "C++ pointer saved at " << allocedPointer << std::endl;
        return (int64_t)allocedPointer;
    }

    void PrintVariable (int64_t allocedPointer_intPtr){
        std::cout << "Reading following value from alloced pointer: " << (*((int*)allocedPointer_intPtr)) << std::endl;
    }

    void DeleteVariable(int64_t allocatedPointer_intPtr){
        std::cout << "Deleting allocated pointer" << std::endl;
        delete (int*)allocatedPointer_intPtr;
        std::cout << "Deleting done" << std::endl;
    }

    // NOTICE: takes a std::map<std::string, std::map<std::string, hipTextureObject_t*>>* cast to an int64_t in order to get around the pybind pointer wierdness.
    void PrintFromFirstTexture (int64_t shaderTextureMaps_Ptr){
        auto shaderTextureMaps = (std::map<std::string, std::map<std::string, hipTextureObject_t*>>*)shaderTextureMaps_Ptr;
        //std::cout << "Cast shaderTextureBundle map pointer from" << shaderTextureMaps_Ptr << " back to " << shaderTextureMaps << std::endl;

        // For each shader, unload all textures from memory
        for(auto shaderTextureBundle : (*shaderTextureMaps)){
            auto textureBundle = shaderTextureBundle.second;

            for(auto texture : textureBundle){
                hipTextureObject_t* texObj = texture.second;
                PrintFirstPixel<<<1,1>>>((*texObj));
                hipDeviceSynchronize();
            }
        }
    }

    // Debug Method used for quickly testing whether 
    __global__ void PrintFirstPixel(hipTextureObject_t texObj){
        float4 cudaTexel = tex2D<float4>(texObj, 0, 0);
        printf("Cuda reading RGBA value of first texel: %f,%f,%f,%f\n", cudaTexel.x, cudaTexel.y, cudaTexel.z, cudaTexel.w);
    }
}