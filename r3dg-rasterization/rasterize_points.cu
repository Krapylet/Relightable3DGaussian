/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include "utils/includeTorch.cu"
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include "cuda_rasterizer/rasterizer_impl.h"
#include <fstream>
#include <string>
#include <functional>
#include "utils/texture.h"
#include "cuda_rasterizer/auxiliary.h"
#include "cuda_rasterizer/postProcessShader.h"

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const float time,
	const float dt,
	const torch::Tensor& means3D, 		//
	const torch::Tensor& features,
    const torch::Tensor& colors,
    const torch::Tensor& opacity, 		//
	const torch::Tensor& scales, 		//
	const torch::Tensor& rotations, 	//
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& shShaderAddresses,
	const torch::Tensor& splatShaderAddresses,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& viewmatrix_inv,
	const torch::Tensor& projmatrix,
	const torch::Tensor& projmatrix_inv,
	const float tan_fovx, 
	const float tan_fovy,
	const float cx,
	const float cy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh, 			
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool computer_pseudo_normal,
	const int64_t d_textureManager_ptr, // is actually a TextureManager* stored on device.
	const std::vector<int64_t> postProcessingPasses_ptr, // is actually a vector of PostProcessShaders
	const bool debug)
{
	// cast the texture manager back into its original class.
	auto d_textureManager = (Texture::TextureManager *const)d_textureManager_ptr;

	// We can't cast the vector to the correct type directly, so we do it in a hacky way instead
	auto ppArray = (PostProcess::PostProcessShader*) &postProcessingPasses_ptr[0];
	auto postProcessingPasses = std::vector<PostProcess::PostProcessShader>(ppArray, ppArray + postProcessingPasses_ptr.size());

	if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
		AT_ERROR("means3D must have dimensions (num_points, 3)");
	}
	
	const int P = means3D.size(0);
	const int S = features.size(1);
	const int H = image_height;
	const int W = image_width;

	auto int_opts = means3D.options().dtype(torch::kInt32);
	auto float_opts = means3D.options().dtype(torch::kFloat32);

	torch::Tensor out_color = torch::full({H, W, NUM_CHANNELS}, 0.0, float_opts);
	torch::Tensor out_opacity = torch::full({H, W, 1}, 0.0, float_opts);
	torch::Tensor out_depth = torch::full({H, W, 1}, 0.0, float_opts);
	torch::Tensor out_stencil = torch::full({H, W, 1}, 0.0, float_opts);
	torch::Tensor out_feature = torch::full({H, W, S}, 0.0, float_opts);
	torch::Tensor out_shader_color = torch::full({H, W, NUM_CHANNELS}, 0.0, float_opts);
	torch::Tensor out_normal = torch::full({H, W, 3}, 0.0, float_opts);
	torch::Tensor out_surface_xyz = torch::full({H, W, 3}, 0.0, float_opts);
	torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
	
	torch::Device device(torch::kCUDA);
	torch::TensorOptions options(torch::kByte);
	torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
	torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
	torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
	std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
	std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
	std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
	//std::cout << "At ShDefault Cracks texture: " << rawTextures.at("ShDefault").at("Cracks") ///  should be .at("rawData")  /// .cpu().contiguous().data_ptr<float>()[0] << ", At ShDefault Red texture:" << rawTextures.at("ShDefault").at("Red").cpu().contiguous().data_ptr<float>()[0] << std::endl;

	// Since the addresses used for these arrays point to the same memory as used in the python frontend, any changes we make to them will stay permanent.
	// While this is an interesting feature (that should maybe be toggleable?) we don't want that right now. We therefore have to copy
	// every array that contains a value we want to be able to change non-persistantly
	torch::Tensor temp_means3D = means3D.detach().clone();
	torch::Tensor temp_features = features.detach().clone();
	torch::Tensor temp_opacity = opacity.detach().clone();
	torch::Tensor temp_scales = scales.detach().clone();
	torch::Tensor temp_rotations = rotations.detach().clone();
	torch::Tensor temp_sh = sh.detach().clone();

	int rendered = 0;
	if(P != 0)
	{
		int M = 0;
		if(sh.size(0) != 0)
		{
			M = sh.size(1);
		}
		rendered = CudaRasterizer::Rasterizer::forward(
			geomFunc,
			binningFunc,
			imgFunc,
			time, dt,
			P, S, degree, M,
			background.contiguous().data_ptr<float>(),
			W, H,
			temp_means3D.contiguous().data_ptr<float>(),
			temp_sh.contiguous().data_ptr<float>(),
			colors.contiguous().data_ptr<float>(),
			temp_features.contiguous().data_ptr<float>(),
			temp_opacity.contiguous().data_ptr<float>(),
			temp_scales.contiguous().data_ptr<float>(),
			scale_modifier,
			temp_rotations.contiguous().data_ptr<float>(),
			cov3D_precomp.contiguous().data_ptr<float>(),
			shShaderAddresses.contiguous().data_ptr<int64_t>(),	
			splatShaderAddresses.contiguous().data_ptr<int64_t>(),
			viewmatrix.contiguous().data_ptr<float>(),
			viewmatrix_inv.contiguous().data_ptr<float>(),
			projmatrix.contiguous().data_ptr<float>(),
			projmatrix_inv.contiguous().data_ptr<float>(),
			campos.contiguous().data_ptr<float>(),
			tan_fovx,
			tan_fovy,
			cx,
			cy,
			prefiltered,
			computer_pseudo_normal,
			d_textureManager,
			postProcessingPasses,
			out_color.contiguous().data_ptr<float>(),
			out_opacity.contiguous().data_ptr<float>(),
			out_depth.contiguous().data_ptr<float>(),
			out_stencil.contiguous().data_ptr<float>(),
			out_feature.contiguous().data_ptr<float>(),
			out_shader_color.contiguous().data_ptr<float>(),
			out_normal.contiguous().data_ptr<float>(),
			out_surface_xyz.contiguous().data_ptr<float>(),
			radii.contiguous().data_ptr<int>(),
			debug);
	}
	char* img_ptr = reinterpret_cast<char*>(imgBuffer.contiguous().data_ptr());
	CudaRasterizer::ImageState imgState = CudaRasterizer::ImageState::fromChunk(img_ptr, H*W);

	torch::Tensor n_contrib = torch::from_blob(imgState.n_contrib, {H, W, 1}, int_opts);
	return std::make_tuple(rendered, n_contrib, out_color, out_opacity, out_depth, out_stencil, out_feature, out_shader_color, out_normal, out_surface_xyz, radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& features,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
    const torch::Tensor& dL_dout_opacity,
    const torch::Tensor& dL_dout_depth,
    const torch::Tensor& dL_dout_feature,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool backward_geometry,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int S = features.size(1);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);

  int M = 0;
  if(sh.size(0) != 0)
  {
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dfeatures = torch::zeros({P, S}, features.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());

  if(P != 0)
  {
	  CudaRasterizer::Rasterizer::backward(P, S, degree, M, R,
	  background.contiguous().data_ptr<float>(),
	  W, H,
	  means3D.contiguous().data_ptr<float>(),
	  sh.contiguous().data_ptr<float>(),
	  features.contiguous().data_ptr<float>(),
	  colors.contiguous().data_ptr<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data_ptr<float>(),
	  viewmatrix.contiguous().data_ptr<float>(),
	  projmatrix.contiguous().data_ptr<float>(),
	  campos.contiguous().data_ptr<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data_ptr<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data_ptr<float>(),
	  dL_dout_opacity.contiguous().data_ptr<float>(),
	  dL_dout_depth.contiguous().data_ptr<float>(),
	  dL_dout_feature.contiguous().data_ptr<float>(),
	  dL_dmeans2D.contiguous().data_ptr<float>(),
	  dL_dconic.contiguous().data_ptr<float>(),
	  dL_dopacity.contiguous().data_ptr<float>(),
	  dL_dcolors.contiguous().data_ptr<float>(),
	  dL_dfeatures.contiguous().data_ptr<float>(),
	  dL_dmeans3D.contiguous().data_ptr<float>(),
	  dL_dcov3D.contiguous().data_ptr<float>(),
	  dL_dsh.contiguous().data_ptr<float>(),
	  dL_dscales.contiguous().data_ptr<float>(),
	  dL_drotations.contiguous().data_ptr<float>(),
	  backward_geometry,
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dfeatures, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data_ptr<float>(),
		viewmatrix.contiguous().data_ptr<float>(),
		projmatrix.contiguous().data_ptr<float>(),
		present.contiguous().data_ptr<bool>());
  }
  
  return present;
}