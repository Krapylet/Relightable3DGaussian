#include "hip/hip_runtime.h"
#include "texture.h"
#include "third_party/lodepng/lodepng.h"
#include "cuda_rasterizer/auxiliary.h"
#include <iostream>
#include <stdexcept>
#include <map>

namespace Texture
{
    __global__ void PrintFirstPixel(hipTextureObject_t texObj){
        //TODO: Wrap a class around the texture object that automatically gets the correct type of value from the texture based on the texture mode.
        float4 cudaTexel = tex2D<float4>(texObj, 0, 0);
        printf("Cuda reading RGBA value of first texel: %f,%f,%f,%f\n", cudaTexel.x, cudaTexel.y, cudaTexel.z, cudaTexel.w);
    }

    // Allocates a new array from the input array where every 4th index is a padded value of 1. The input pointer is overwritten with the pointer to the new array.
    // The array is 4/3rds the length of the input array. Remember to delete the allocated array.
    __global__ void CreatPaddedArrayFromBase(float* src, float* dest, int width, int height){
        // Increase count by 1/3rd to make room for the 4th channel
        for (size_t padded_i = 0; padded_i < width*height*4; padded_i++)
        {
            int i = 0;
            bool padCurrentIndex = (padded_i + 1) % 4 == 0;
            if (padCurrentIndex)
            {
                dest[padded_i] = 1;
            }
            else{
                dest[padded_i] = src[i];
                i++;
            }
        }
    }

    /*
    Create a channel descriptor appropriate to the image mode.
    Pillow supported channel modes:
    1 (1-bit pixels, black and white, stored with one pixel per byte)
    L (8-bit pixels, grayscale)
    P (8-bit pixels, mapped to any other mode using a color palette)
    RGB (3x8-bit pixels, true color)
    RGBA (4x8-bit pixels, true color with transparency mask)
    CMYK (4x8-bit pixels, color separation)
    YCbCr (3x8-bit pixels, color video format)
    Note that this refers to the JPEG, and not the ITU-R BT.2020, standard
    LAB (3x8-bit pixels, the L*a*b color space)
    HSV (3x8-bit pixels, Hue, Saturation, Value color space)
    Hue’s range of 0-255 is a scaled version of 0 degrees <= Hue < 360 degrees
    I (32-bit signed integer pixels)
    F (32-bit floating point pixels)
    https://pillow.readthedocs.io/en/stable/handbook/concepts.html
    */
    int encodeTextureMode(std::string mode){
        if(mode == "1")
            return TextureMode::One;
        else if ( mode == "L")
            return TextureMode::L;
        else if ( mode == "P")
            return TextureMode::P;
        else if ( mode == "RGB")
            return TextureMode::RGB;
        else if ( mode == "RGBA")
            return TextureMode::RGBA;
        else if ( mode == "CMYK")
            return TextureMode::CMYK;
        else if ( mode == "YCbCr")
            return TextureMode::YCbCr;
        else if ( mode == "LAB")
            return TextureMode::LAB;
        else if ( mode == "HSV")
            return TextureMode::HSV;
        else if ( mode == "I")
            return TextureMode::I;
        else if ( mode == "F")
            return TextureMode::F;

        return TextureMode::Unknown;
    }

    // Creates a textureObject wrapper around the provided texture data
    // Adapted from the lodepng decoding example example at https://github.com/lvandeve/lodepng/blob/master/examples/example_decode.cpp
    // and the cuda example at https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api
    void CreateTexture(hipTextureObject_t* texObjPtr, std::map<std::string, torch::Tensor> textureData){
        // extract all the texture data
        int height = textureData["height"].const_data_ptr<int>()[0];
        int width = textureData["width"].const_data_ptr<int>()[0];
        TextureMode mode = static_cast<TextureMode>(textureData["mode"].const_data_ptr<int>()[0]);
        float* pixelData = textureData["pixelData"].contiguous().cuda().mutable_data_ptr<float>();
        int pixelDataCount = textureData["pixelData"].numel();

        /*
        Create a channel descriptor appropriate to the image mode.
        Pillow supported channel modes:
        1 (1-bit pixels, black and white, stored with one pixel per byte)
        L (8-bit pixels, grayscale)
        P (8-bit pixels, mapped to any other mode using a color palette)
        RGB (3x8-bit pixels, true color)
        RGBA (4x8-bit pixels, true color with transparency mask)
        CMYK (4x8-bit pixels, color separation)
        YCbCr (3x8-bit pixels, color video format)
        Note that this refers to the JPEG, and not the ITU-R BT.2020, standard
        LAB (3x8-bit pixels, the L*a*b color space)
        HSV (3x8-bit pixels, Hue, Saturation, Value color space)
        Hue’s range of 0-255 is a scaled version of 0 degrees <= Hue < 360 degrees
        I (32-bit signed integer pixels)
        F (32-bit floating point pixels)
        https://pillow.readthedocs.io/en/stable/handbook/concepts.html
        */
        hipChannelFormatDesc channelDesc;
        int paddedDataSize;
        int channelByteWidth;
        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        if(mode == TextureMode::One || mode == TextureMode::L || mode == TextureMode::P || mode == TextureMode::I || mode == TextureMode::F){
            channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float1);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGBA || mode == TextureMode::CMYK)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3, so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            CreatPaddedArrayFromBase<<<1,1>>>(pixelData, paddedData, width, height);         //TODO: accelerate this kernel with more threads 
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        else if (mode == TextureMode::LAB|| mode == TextureMode::HSV)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModePoint;  // Linear filtering is only available for floats 
            texDesc.readMode = hipReadModeNormalizedFloat; //Notice: This might actually not work with 32 bit channels. Only 16 and 8 bit channels. But I haven't tested it.

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3m so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            CreatPaddedArrayFromBase<<<1,1>>>(pixelData, paddedData, width, height);         //TODO: accelerate this kernel with more threads 
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        // Specify remaining texture object parameters
        texDesc.addressMode[0] = hipAddressModeWrap;
        texDesc.addressMode[1] = hipAddressModeWrap;         //TODO: allow the wrap mode to be set based on a texture import setting
        texDesc.normalizedCoords = 1;                           //TODO: allow the coordinate mode ot be set based on a texture import setting

        // Allocate CUDA array in device memory
        hipArray_t cuArray;
        hipMallocArray(&cuArray, &channelDesc, width, height);

        // Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src, including padding). We dont have any padding, so it's just equal to the byte width.
        const size_t pitch = channelByteWidth;
        checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, pixelData, pitch, channelByteWidth, height, hipMemcpyDeviceToDevice));

        // Specify texture
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;
        
        // Create texture object, which is used as a wrapper to access the cuda Array with the actual image data.
        checkCudaErrors(hipCreateTextureObject(texObjPtr, &resDesc, &texDesc, NULL));
        
        if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr || mode == TextureMode::LAB || mode == TextureMode::HSV)
        {   
            // If we had to copy and pad the data of a 3-value format with a 4th value before the data was copied to a hipArray,
            // we have to free the memory used to create the temporary padded version. 
            hipFree(pixelData);
        }
 
        // TODO: Make sure to keep track of which memory we need clean up at the end of this function, and at the end of this frame.
    }

    // TODO:unload texture
    // Free device memory at the end of the frame
    // hipResourceDesc* resDesc_DeleteThisPart;
    // hipGetTextureObjectResourceDesc(resDesc_DeleteThisPart, texObj);
    //hipFreeArray(resDesc_DeleteThisPart->res.array.array);
}



