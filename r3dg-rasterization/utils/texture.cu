#include "hip/hip_runtime.h"
#include "texture.h"
#include "../cuda_rasterizer/auxiliary.h"
#include <iostream>
#include <stdexcept>
#include <map>
#include <hip/hip_cooperative_groups.h>
#include "charOperations.h"


namespace Texture
{
    // Allocates a new array from the input array where every 4th index is a padded value of 1. The input pointer is overwritten with the pointer to the new array.
    // The array is 4/3rds the length of the input array. Remember to delete the allocated array.
    __global__ void CreatPaddedArrayFromBase(float* src, float* dest, int paddedDataCount){
        auto padded_i = cooperative_groups::this_grid().thread_rank();
        if (padded_i >= paddedDataCount)
            return;

        bool padCurrentIndex = (padded_i + 1) % 4 == 0;
        if (padCurrentIndex)
        {
            dest[padded_i] = 1;
        }
        else{
            // paddedDataCount = height * width * 4
            // srcDataCount = height * width * 3
            int src_i = ceil(padded_i*0.75);
            dest[padded_i] = src[src_i];
        }
    }

    // Encodes a string representing a texture mode to a TextureMode enum
    int EncodeTextureMode(std::string mode){
        if(mode == "1")
            return TextureMode::One;
        else if ( mode == "L")
            return TextureMode::L;
        else if ( mode == "P")
            return TextureMode::P;
        else if ( mode == "RGB")
            return TextureMode::RGB;
        else if ( mode == "RGBA")
            return TextureMode::RGBA;
        else if ( mode == "CMYK")
            return TextureMode::CMYK;
        else if ( mode == "YCbCr")
            return TextureMode::YCbCr;
        else if ( mode == "LAB")
            return TextureMode::LAB;
        else if ( mode == "HSV")
            return TextureMode::HSV;
        else if ( mode == "I")
            return TextureMode::I;
        else if ( mode == "F")
            return TextureMode::F;

        return TextureMode::Unknown;
    }

    // Encodes string to cuda enum. Possible modes are:
    // - "Wrap": UVs outside the range wraps back around from the other side, repeating the texture.
    // - "Mirror": UBs outside the range wraps back from the same side, repeating the texture, but mirrored.
    // - "Clamp": UVs outside the range are clamped back into the range of the texture
    // - "Border": UVs outside the range return 0.
    int EncodeWrapMode(std::string mode){
        if(mode == "Border")
            return hipAddressModeBorder;
        if(mode == "Clamp")
            return hipAddressModeClamp;
        if(mode == "Mirror")
            return hipAddressModeMirror;
        if(mode == "Wrap")
            return hipAddressModeWrap;

        return -1;
    }

    // NOTICE: Returns a hipTextureObject_t* cast to a int64_t
    // Creates a textureObject wrapper around the provided texture data and allocates it in memory.
    // Used in order to intialize textures outside of render loop.
    int64_t AllocateTexture(std::map<std::string, torch::Tensor> textureData){
        hipTextureObject_t* texObj = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t));

        CreateTexture(texObj, textureData);

        return (int64_t)texObj;
    }

    // Creates a textureObject wrapper around the provided texture data
    // Adapted from the cuda example at https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api
    void CreateTexture(hipTextureObject_t* texObjPtr, std::map<std::string, torch::Tensor> textureData){
        // extract all the texture data
        int height = textureData["height"].const_data_ptr<int>()[0];
        int width = textureData["width"].const_data_ptr<int>()[0];
        TextureMode mode = static_cast<TextureMode>(textureData["encoding_mode"].const_data_ptr<int>()[0]);
        float* pixelData = textureData["pixelData"].contiguous().cuda().mutable_data_ptr<float>();
        int pixelDataCount = textureData["pixelData"].numel();
        bool normalizedCoords = textureData["normalizedCoords"].const_data_ptr<int>()[0];
        const hipTextureAddressMode* addressModes = (const hipTextureAddressMode*)textureData["wrap_modes"].const_data_ptr<int>();

        /*
        Create a channel descriptor appropriate to the image mode.
        Pillow supported channel modes:
        1 (1-bit pixels, black and white, stored with one pixel per byte)
        L (8-bit pixels, grayscale)
        P (8-bit pixels, mapped to any other mode using a color palette)
        RGB (3x8-bit pixels, true color)
        RGBA (4x8-bit pixels, true color with transparency mask)
        CMYK (4x8-bit pixels, color separation)
        YCbCr (3x8-bit pixels, color video format)
        Note that this refers to the JPEG, and not the ITU-R BT.2020, standard
        LAB (3x8-bit pixels, the L*a*b color space)
        HSV (3x8-bit pixels, Hue, Saturation, Value color space)
        Hue’s range of 0-255 is a scaled version of 0 degrees <= Hue < 360 degrees
        I (32-bit signed integer pixels)
        F (32-bit floating point pixels)
        https://pillow.readthedocs.io/en/stable/handbook/concepts.html
        */
        hipChannelFormatDesc channelDesc;
        int paddedDataSize;
        int channelByteWidth;
        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        if(mode == TextureMode::One || mode == TextureMode::L || mode == TextureMode::P || mode == TextureMode::I || mode == TextureMode::F){
            channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float1);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGBA || mode == TextureMode::CMYK)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;
        }
        else if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModeLinear;
            texDesc.readMode = hipReadModeElementType;

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3, so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        else if (mode == TextureMode::LAB|| mode == TextureMode::HSV)
        {
            channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
            channelByteWidth = width * sizeof(float4);
            paddedDataSize = height * channelByteWidth;
            texDesc.filterMode = hipFilterModePoint;  // Linear filtering is only available for floats 
            texDesc.readMode = hipReadModeNormalizedFloat; //Notice: This might actually not work with 32 bit channels. Only 16 and 8 bit channels. But I haven't tested it.

            // CUDA only support textures with 1,2 or 4 channels pr. pixel, not 3m so we have to pad it with an additional value. In this case I'm just adding a 4th opaque alpha channel.
            float* paddedData;
            hipMalloc(&paddedData, paddedDataSize);
            int paddedDataCount = width*height*4;
            CreatPaddedArrayFromBase<<<(paddedDataCount + 255) / 256, 256>>>(pixelData, paddedData, paddedDataCount);
            hipDeviceSynchronize();
            pixelData = paddedData; // Overwrite the original data pointer. Remember to free the memory by the end of the function.
        }
        // Specify remaining texture object parameters
        texDesc.addressMode[0] = addressModes[0];
        texDesc.addressMode[1] = addressModes[1];
        texDesc.normalizedCoords = normalizedCoords;

        // Allocate CUDA array in device memory
        hipArray_t cuArray;
        hipMallocArray(&cuArray, &channelDesc, width, height);

        // Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src, including padding). We dont have any padding, so it's just equal to the byte width.
        const size_t pitch = channelByteWidth;
        checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, pixelData, pitch, channelByteWidth, height, hipMemcpyDeviceToDevice));

        // Specify texture
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;
        
        // Create texture object, which is used as a wrapper to access the cuda Array with the actual image data.
        checkCudaErrors(hipCreateTextureObject(texObjPtr, &resDesc, &texDesc, NULL));

        if ( mode == TextureMode::RGB || mode == TextureMode::YCbCr || mode == TextureMode::LAB || mode == TextureMode::HSV)
        {   
            // If we had to copy and pad the data of a 3-value format with a 4th value before the data was copied to a hipArray,
            // we have to free the memory used to create the temporary padded version. 
            hipFree(pixelData);
        }
    }

    // Frees the underlying hipArray that the textureObject is wrapped around, as well as the texture object pointer that contains it.
    void UnloadTexture(hipTextureObject_t* textureObject){
        hipResourceDesc resDesc;
        checkCudaErrors(hipGetTextureObjectResourceDesc(&resDesc, (*textureObject)));
        checkCudaErrors(hipFreeArray(resDesc.res.array.array));
    }

    // Frees the underlying hipArray that the textureObject is wrapped around
    void UnloadTexture(hipTextureObject_t textureObject){
        hipResourceDesc resDesc;
        checkCudaErrors(hipGetTextureObjectResourceDesc(&resDesc, textureObject));
        checkCudaErrors(hipFreeArray(resDesc.res.array.array));
    }

    // initialize device texture vector and device texture name vector (used for indirect addressing of textures)
    // NOTICE: actually returns a d_TextureManager* cast to an int64.
    int64_t UploadTexturesToDevice(std::vector<std::string> names, std::vector<int64_t> textureObjects, int64_t errorTexture){
        auto h_texManager = new TextureManager();
        h_texManager->SetTextures(names, textureObjects);
        h_texManager->SetErrorTexture((hipTextureObject_t*) errorTexture);

        TextureManager* d_texManager;
        checkCudaErrors(hipMalloc(&d_texManager, sizeof(TextureManager)));
        checkCudaErrors(hipMemcpy(d_texManager, h_texManager, sizeof(TextureManager), hipMemcpyHostToDevice));

        return (int64_t)d_texManager;
    }
}

/// -----------------------Texture manager class implementation -------------------

    __host__ Texture::TextureManager::TextureManager(){};

    // Allocates and Uploads an array  of textures onto the GPU so that textures can be looked up by the shaders.
    __host__ void Texture::TextureManager::SetTextures(std::vector<std::string> names, std::vector<int64_t> textureObjects){
        // First, move each element into a vector on host
        // We use vectors instead of arrays since we don't know the size at compile time, and we don't wanna allocate memory ourselves.
        int h_texCount = names.size();
        std::vector<char*> h_names (h_texCount);
        std::vector<hipTextureObject_t> h_texObjs(h_texCount);

        for (size_t i = 0; i < h_texCount; i++)
        {
            std::string name = names[i];
            hipTextureObject_t texObj = *((hipTextureObject_t*)textureObjects[i]);

            // convert each name to a char array located in device memory
            // Texture objects are already in device memory, so we don't need to do anything to them.
            int stringlength = name.length();
            char* d_charName;
            hipMalloc(&d_charName, (stringlength+1)*sizeof(char)); // add 1 to also include the termination character
            hipMemcpy(d_charName, name.c_str(), (stringlength+1)*sizeof(char), hipMemcpyKind::hipMemcpyDefault);

            // Save the pointers to the tempoary host vectors.
            h_names[i] = d_charName;
            h_texObjs[i] = texObj;
        }
        // Then allocate all the arrays on the device and transfer the data stored on host to them (and to the texCount variable):
        hipMalloc(&d_texCount, sizeof(int));
        hipMemcpy(d_texCount, &h_texCount, sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&d_textureNames, h_texCount * sizeof(char*));
        hipMemcpy(d_textureNames, &h_names[0], h_texCount *sizeof(char*), hipMemcpyHostToDevice);

        hipMalloc(&d_textureObjects, h_texObjs.size() * sizeof(hipTextureObject_t));
        hipMemcpy(d_textureObjects, &h_texObjs[0], h_texObjs.size() * sizeof(hipTextureObject_t), hipMemcpyKind::hipMemcpyDefault);
    }
            
    // Deallocates all textures on the device (except the error texture)
    __host__ void Texture::TextureManager::UnloadTextures(){
        for (size_t i = 0; i < *d_texCount; i++)
        {
            char* name = d_textureNames[i];
            hipTextureObject_t texObj = d_textureObjects[i];

            delete(name);
            UnloadTexture(texObj);
        }
    }

    // Allocates and uploads an error texture to the device.
    __host__ void Texture::TextureManager::SetErrorTexture(hipTextureObject_t* errorTexture){
        checkCudaErrors(hipMalloc(&d_errorTexture, sizeof(hipTextureObject_t)));
        checkCudaErrors(hipMemcpy(&d_errorTexture, errorTexture, sizeof(hipTextureObject_t), hipMemcpyKind::hipMemcpyDefault));
    }

    //Deallocates the error texture on the device.
    __host__ void Texture::TextureManager::UnloadErrorTexture(){
        UnloadTexture(d_errorTexture);
    }

    // Returns the error texture.
    __device__ hipTextureObject_t Texture::TextureManager::GetErrorTexture() const {return *d_errorTexture;}
            
    // Loops through each loaded texture name and checks if it matches the given name before returning the associated texture. Is pretty slow, so cache the result.
    // Returns the error texture if no texture is found.
    __device__ hipTextureObject_t Texture::TextureManager::GetTexture(char* targetTextureName) const {
        for (size_t i = 0; i < *d_texCount; i++)
        {
            // Check if the name in the lookup table is the same as the target name
            char* currentTexName = d_textureNames[i];
            bool textureHasBeenFound = charsAreEqual(currentTexName, targetTextureName);
            
            if(textureHasBeenFound){
                hipTextureObject_t texObj = d_textureObjects[i];
                return texObj;
            }
        }

        // If no texture was found with the given name, return the default error texture
        printf("Warning: Could not find texture '%s'\n", targetTextureName);
        return *d_errorTexture;
    }

    